#include "hip/hip_runtime.h"
#include "nbody_gpu.h"

__global__
void bodyForce_v1(Body *p, float dt, int n) {
  int i_id = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_i = gridDim.x * blockDim.x;
  
  for (int i = i_id; i < n; i+=stride_i) {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].pos.x - p[i].pos.x;
      float dy = p[j].pos.y - p[i].pos.y;
      float dz = p[j].pos.z - p[i].pos.z;
      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;      

      Fx += dx * invDist3; 
      Fy += dy * invDist3; 
      Fz += dz * invDist3;
    }
    
    p[i].vel.x += dt*Fx; 
    p[i].vel.y += dt*Fy; 
    p[i].vel.z += dt*Fz;
  }
}

__global__
void integrateBody(Body *p, float dt, int n){
  int i_id = blockIdx.x * blockDim.x + threadIdx.x;
  int stride_i = gridDim.x * blockDim.x;
  
  for (int i = i_id; i < n; i+=stride_i) {
    p[i].pos.x += p[i].vel.x*dt;
    p[i].pos.y += p[i].vel.y*dt;
    p[i].pos.z += p[i].vel.z*dt;
  }
}

__device__ 
float3 bodyBodyInteraction(float3 ai, float3 bi, float3 bj) {
    float3 r;

    r.x = bi.x - bj.x;
    r.y = bi.y - bj.y;
    r.z = bi.z - bj.z;
    float distSqr = r.x*r.x + r.y*r.y + r.z*r.z + SOFTENING;
    float invDist = rsqrtf(distSqr);
    float invDist3 = invDist * invDist * invDist;      

    ai.x += r.x * invDist3;
    ai.y += r.y * invDist3;
    ai.z += r.z * invDist3;

    return ai;
}

__device__ 
float3 gravitation(float3 myPos, float3 accel)
{
    extern __shared__ float3 sharedPos[];
    int i;

    for (i = 0; i < blockDim.x; ) 
    {
        accel = bodyBodyInteraction(accel, SX(i), myPos); i += 1;
        // Here we unroll the loop if needed
        //accel = bodyBodyInteraction(accel, SX(i), myPos); i += 1;
        //accel = bodyBodyInteraction(accel, SX(i), myPos); i += 1;
        //accel = bodyBodyInteraction(accel, SX(i), myPos); i += 1;
    }

    return accel;
}

template <bool multithreadBodies>
__device__
float3 bodyForceSM(float3 pos, Body *p, float dt, int n) {
  extern __shared__ float3 sharedPos[];
  
  float3 acc = {0.0f, 0.0f, 0.0f};
  
  for (int i = 0; i < gridDim.x; i++) {
    auto data = p[i*blockDim.x + threadIdx.x];
    sharedPos[threadIdx.x+blockDim.x*threadIdx.y] = data.pos;
    __syncthreads();

    // This is the "tile_calculation" function from the GPUG3 article.
    acc = gravitation(pos, acc);
    __syncthreads();
    
  }

  if (multithreadBodies)
  {
      SX_SUM(threadIdx.x, threadIdx.y) = acc;

      __syncthreads();

      // Save the result in global memory for the integration step
      if (threadIdx.y == 0) {
          for (int i = 1; i < blockDim.y; i++) {
              acc.x += SX_SUM(threadIdx.x,i).x;
              acc.y += SX_SUM(threadIdx.x,i).y;
              acc.z += SX_SUM(threadIdx.x,i).z;
          }
      }
  }

  return acc;
}

template <bool multithreadBodies>
__global__
void integrateBodySM(Body *p, float dt, int n){
  int i_id = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (i_id < n){
    Body bodyCurrent = p[i_id];   
    float3 pos = bodyCurrent.pos;
    float3 vel = bodyCurrent.vel;
    
    float3 force = bodyForceSM<multithreadBodies>(pos, p, dt, n);    
    vel.x += force.x * dt;
    vel.y += force.y * dt;
    vel.z += force.z * dt;  
        
    // new position = old position + velocity * deltaTime
    pos.x += vel.x * dt;
    pos.y += vel.y * dt;
    pos.z += vel.z * dt;

    // store new position and velocity
    p[i_id].pos = pos;
    p[i_id].vel = vel;
  }
}

template __device__ float3 bodyForceSM<false>(float3 pos, Body *p, float dt, int n);
template __device__ float3 bodyForceSM<true>(float3 pos, Body *p, float dt, int n);
template __global__ void integrateBodySM<false>(Body *p, float dt, int n);
template __global__ void integrateBodySM<true>(Body *p, float dt, int n);